#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset

  // const size_t tid = threadIdx.x + blockDim.x * threadIdx.y;

  const size_t tid = blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x)  + threadIdx.x;

  const size_t tid_max = numRows * numCols - 1;
  if (tid > tid_max)
  {
    return;
  }


  // std::cout << "blockIdx.x: " << blockIdx.x << std::endl;
  // printf("INDICES: %d, %d) %d, %d | %d \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, tid);

  const char R = rgbaImage[tid].x;
  const char G = rgbaImage[tid].y;
  const char B = rgbaImage[tid].z;

  greyImage[tid] = .299f * (int)R + .587f * (int)G + .114f * (int)B;
  // greyImage[tid] = (unsigned char)(.299f * (int)R + .587f * (int)G + .114f * (int)B);
  // greyImage[tid] = (unsigned char)(.299f * (uint8_t)R + .587f * (uint8_t)G + .114f * (uint8_t)B);
 
  printf("val: %c, %c, %c | %d, %d, %d \n", R, G, B, (int)R, (int)G, (int)B);
  // printf("val: %c, %c, %c | %d, %d, %d \n", R, G, B, R, G, B);
  // printf("val: %d) %c, %c, %c | %d, %d, %d \n", tid, R, G, B, (uint8_t)R, (uint8_t)G, (uint8_t)B);


}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  /** 
   * Notes about dimensions: 
   * 
   * Going to use 1 block that is the dimension of the image in 2D. ie. Different thread per pixel.
   * 
   * Block does not require depth/dimension corresponding to image channels per pixel, since the 
   * algorithm to convert a pixel to greyscale uses all three/four channels per pixels and is not dependent on the depth
   * per pixel. ie. No iteration index required along channels of each pixel.
   * 
   * A single grid with the single block is sufficient.
   */
  // const dim3 blockSize(numRows, numCols, 1); // ie. Threads per block
  // const dim3 gridSize(1, 1, 1); // ie. Blocks per grid
  // rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);


  // TODO: Is there a way to get this max value at run time than hardcoding it so that all dims of
  // block and grid can be set w.r.t that max value to avoid breaching max limits?
  // const size_t block_max_dim = 1024;


  /** 
   * Follow up notes about dimensions: 
   * Run "/usr/local/cuda/extras/demo_suite/deviceQuery" on command line to get device hardware configs
   * 
   * Max threads per block is 1024. This means multiple blocks are necessary. 
   * Also, max size per block in each dimension is (1024, 1024, 64)
   * 
   * We have 313 * 557 = 174,341 pixels to process
   * So, if a block can process 1024 pixels, we need at least 174,341 / 1024 = 170.25 = 171 blocks
   * So, let a grid of blocks of 171 be created. Can further reduce this to a square of blocks if necessary, 
   * but 171 falls under the max size of each dimension of a grid so will proceed as a single dimension grid.
   * 
   * NOTE: Not all threads in the last block will be necessary, so a conditional check will be necessary to 
   * return if total thread index becomes > total number of pixels.
   */
  const dim3 blockSize(32, 32, 1); // 1024 threads per block
  const dim3 gridSize(171, 1, 1); // 256 blocks per grid
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
